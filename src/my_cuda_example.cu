#include "hip/hip_runtime.h"
#include "my_cuda_example.h"

/*************************************************************/
__device__ int my_kernel_helper(int a, int b)
{
    return a*b;
}
/*************************************************************/
__global__ void my_kernel()
{
    // extern __shared__ int my_dyn_shared_mem[];
    //        __shared__ int my_stat_shared_mem[42];
    //        __shared__ int shared_var;
    //
    // blockIdx.x
    // blockDim.x
    // threadIdx.x
    //
    // __syncthreads();
    //
    // atomicAdd(&shared_var, 167);
}
/*************************************************************/
void run_on_cuda()
{
    float* device_pointer;
    int size = 1024*sizeof(float);

    hipMalloc(&device_pointer, size);
    //
    // hipMemcpy(device_pointer, host_pointer, size, hipMemcpyHostToDevice);
    //
    // my_kernel<<<a, b, c>>>();
    //
    // hipMemcpy(host_pointer, device_pointer, size, hipMemcpyDeviceToHost);
    //
    hipFree(device_pointer);
}
/*************************************************************/
