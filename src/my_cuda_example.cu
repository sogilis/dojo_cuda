#include "hip/hip_runtime.h"
#include "my_cuda_example.h"
#include <ctime>
#include <stdio.h>

typedef struct
{
    float speed_x, speed_y;
    float pos_x, pos_y;
} particle;
/*************************************************************/
__global__ void my_kernel(particle* particles)
{
    particle *p = &particles[blockIdx.x*blockDim.x + threadIdx.x];
    p->pos_x += p->speed_x;
    p->pos_y += p->speed_y;
}
/*************************************************************/
void run_on_cuda()
{
    int n = 1024*1024*100;

    particle* host_pointer, *device_pointer;
    int size = n*sizeof(particle);

    //allocation
    host_pointer = (particle*)malloc(size);
    hipMalloc(&device_pointer, size);

    //fill host data
    //srand(time(NULL));
    for(int i=0 ; i<n ; i++)
    {
        host_pointer[i].speed_x = 1.0;
        host_pointer[i].speed_y = 3.2;
        host_pointer[i].pos_x = 0.0;
        host_pointer[i].pos_y = 0.0;
    }

    //send data
    hipMemcpy(device_pointer, host_pointer, size, hipMemcpyHostToDevice);

    //compute
    #define CUDA_EXEC
    clock_t t = clock();
   
    #ifdef CUDA_EXEC
        my_kernel<<<n / 1024, 1024>>>(device_pointer);
    #else
        for(int i=0 ; i<n ; i++)
        {
            particle *p = &host_pointer[i];
            p->pos_x += p->speed_x;
            p->pos_y += p->speed_y;
        }
    #endif
    clock_t t2 = clock() - t;
    printf("%f secs\n", (double)t2 / CLOCKS_PER_SEC);
    //retrieve data
    hipMemcpy(host_pointer, device_pointer, size, hipMemcpyDeviceToHost);

    hipFree(device_pointer);
    free(host_pointer);
}
/*************************************************************/
